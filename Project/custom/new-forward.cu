#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int w_out = threadIdx.x + blockIdx.x * blockDim.x;
    int h_out = threadIdx.y + blockIdx.y * blockDim.y;
    int m = threadIdx.z + blockIdx.z * blockDim.z;

    if (w_out < W_out && h_out < H_out && m < M) {
        for (int b = 0; b < B; ++b) {
            float value = 0.0f;
            for (int c = 0; c < C; ++c) {
                for (int p = 0; p < K; ++p) {
                    for (int q = 0; q < K; ++q) {
                        int h_in = h_out * S + p;
                        int w_in = w_out * S + q;
                        value += in_4d(b, c, h_in, w_in) * mask_4d(m, c, p, q);
                    }
                }
            }
            out_4d(b, m, h_out, w_out) = value;
        }
    }


    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    // Allocate memory for device_input, device_output, device_mask
    hipMalloc(device_input_ptr, B * C * H * W * sizeof(float));
    hipMalloc(device_output_ptr, B * M * ((H - K) / S + 1) * ((W - K) / S + 1) * sizeof(float));
    hipMalloc(device_mask_ptr, M * C * K * K * sizeof(float));
    
    // Copy data from host_input, host_output, host_mask to device
    hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    
    // Initialize device_output to zero
    hipMemset(*device_output_ptr, 0, B * M * ((H - K) / S + 1) * ((W - K) / S + 1) * sizeof(float));
   
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
       // Calculate grid and block sizes
    const int H_out = (H - K) / S + 1;
    const int W_out = (W - K) / S + 1;

    dim3 blockDim(16, 16, 4);
    dim3 gridDim((W_out + blockDim.x - 1) / blockDim.x, (H_out + blockDim.y - 1) / blockDim.y, (M + blockDim.z - 1) / blockDim.z);
    
    // Launch the conv_forward_kernel
    conv_forward_kernel<<<gridDim, blockDim>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);
    
    // Check for errors after kernel launch
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy output from device to host_output
    hipMemcpy(host_output, device_output, B * M * ((H - K) / S + 1) * ((W - K) / S + 1) * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory for device_input, device_output, device_mask
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
    
    // Check for errors after copy and free
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
